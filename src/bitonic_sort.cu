/*
  Erick Wijaya / 13515057
  source: https://www2.cs.duke.edu/courses/fall08/cps196.1/Pthreads/bitonic.c
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <string.h>
#include <math.h>

const int ASCENDING  = 1;
const int DESCENDING = 0;
const int MAX_INT = 2147483647;
const int THREADS = 4;

void init(int* arr, int n);
void rng(int* arr, int n);
void print(int* arr);
void test(int* arr, int n);
void swap(int* a, int* b);
void compare(int* arr, int i, int j, int dir);
void bitonicSortSeq(int* arr, int n);

int nearestPowerOfTwo(int x);
void writeToFile(char* filename, int* arr, int n);

/** the main program **/ 
int main(int argc, char **argv) {
  int* arr;
  int i, j;
  int N, fakeN;
  int num_thread;

  struct timeval startwtime, endwtime;
  double seq_time;
  FILE* log_file;

  double sum_parallel = 0;

  if (argc == 2) {
    // pass
    num_thread = THREADS;
  }
  else if (argc == 3) {
    num_thread = atoi(argv[2]);
  }
  else {
    printf("Usage: %s n x\n  where n is problem size and p is thread count\n", argv[0]);
    return 0;
  }
  
 
  // Initialize arr
  N = atoi(argv[1]);
  fakeN = nearestPowerOfTwo(N);
  arr = (int *) malloc(fakeN * sizeof(int));
  init(arr, fakeN);
  rng(arr, N);

  writeToFile("data/input.txt", arr, N);
  log_file = fopen("output/log.txt", "a");
  if (log_file == NULL) {
    printf("Error: can't open/create file");
    exit(1);
  }

  time_t rawtime;
  struct tm* timeinfo;
  time(&rawtime);
  timeinfo = localtime(&rawtime);
  printf("-----------------------------------------\n");
  printf("%s", asctime(timeinfo));
  printf("Problem Size: %d\n", N);
  printf("Process: %d\n", num_thread);
  fprintf(log_file, "-----------------------------------------\n");
  fprintf(log_file, "%s", asctime(timeinfo));
  fprintf(log_file, "Problem Size: %d\n", N);
  fprintf(log_file, "Process: %d\n", num_thread);

  int* newArr;
  newArr = (int *) malloc(fakeN * sizeof(int));
  memcpy(newArr, arr, fakeN);

  // [Start Time]
  gettimeofday (&startwtime, NULL);

  bitonicSortSeq(newArr, fakeN);

  gettimeofday (&endwtime, NULL);
  // [End Time]


  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);
  seq_time *= 1000000;

  printf("Parallel wall clock time (microseconds) = %f\n", seq_time);
  fprintf(log_file, "Parallel wall clock time (microseconds) = %f\n", seq_time);

  writeToFile("data/output.txt", newArr, N);
      
  test(newArr, fakeN);
  sum_parallel += seq_time;
  free(newArr);
  

  printf("-----------------------------------------\n");
  fprintf(log_file, "-----------------------------------------\n\n\n");
  fclose(log_file);
  

  return 0;
}

/** -------------- SUB-PROCEDURES  ----------------- **/ 
void init(int* arr, int n) {
  int i = 0;
  for (i = 0; i < n; i++) {
    arr[i] = MAX_INT;
  }
}

void writeToFile(char* filename, int* arr, int n) {
  FILE* write = fopen(filename, "w");
  if (write != NULL) {
    int j;
    for (j = 0; j < n; j++)
      fprintf(write, "%d\n", arr[j]);
    fclose(write);
  }
}

void test(int* arr, int n) {
  int pass = 1;
  int i;
  for (i = 1; i < n; i++) {
    pass &= (arr[i-1] <= arr[i]);
  }

  printf("TEST %s\n",(pass) ? "PASSED" : "FAILED");
}

void rng(int* arr, int n) {
  int seed = 13515057;
  srand(seed);
  int i;
  for (i = 0; i < n; i++) {
    arr[i] = (int)rand();
  }
}

void swap(int* a, int* b) {
  int t;
  t = *a;
  *a = *b;
  *b = t;
}

void compare(int* arr, int i, int j, int dir) {
  if (dir == (arr[i] > arr[j])) 
    swap(&(arr[i]), &(arr[j]));
}

void bitonicSortSeq(int* arr, int n) {
  int i,j,k;
  
  for (k=2; k<=n; k=2*k) {
    for (j=k>>1; j>0; j=j>>1) {
      for (i=0; i<n; i++) {
	      int ij=i^j;
        if ((ij) > i) {
          if ((i&k) == 0 && arr[i] > arr[ij]) 
            swap(&(arr[i]), &(arr[ij]));
          if ((i&k) != 0 && arr[i] < arr[ij])
            swap(&(arr[i]), &(arr[ij]));
        }
      }
    }
  }
}

int nearestPowerOfTwo(int x) {
  int i = 2;
  while (i < x) {
    i *= 2;
  }

  return i;
}
