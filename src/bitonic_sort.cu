#include "hip/hip_runtime.h"
/*
  Erick Wijaya / 13515057
  source: https://www2.cs.duke.edu/courses/fall08/cps196.1/Pthreads/bitonic.c
*/

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <string.h>
#include <math.h>

const int ASCENDING  = 1;
const int DESCENDING = 0;
const int MAX_INT = 2147483647;

int THREADS;
int BLOCKS;
int NUM_VALS;

void init(int* arr, int n);
void rng(int* arr, int n);
void print(int* arr);
void test(int* arr, int n);
void swap(int* a, int* b);
void compare(int* arr, int i, int j, int dir);
void parBitonicSort(int* arr);

int nearestPowerOfTwo(int x);
void writeToFile(char* filename, int* arr, int n);

__global__ void bitonicSortStep(int* arr, int j, int k) {
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (arr[i]>arr[ixj]) {
        /* exchange(i,ixj); */
        int temp = arr[i];
        arr[i] = arr[ixj];
        arr[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (arr[i]<arr[ixj]) {
        /* exchange(i,ixj); */
        int temp = arr[i];
        arr[i] = arr[ixj];
        arr[ixj] = temp;
      }
    }
  }
}

/** the main program **/ 
int main(int argc, char **argv) {
  int* arr;
  int i, j;
  int N, fakeN;

  struct timeval startwtime, endwtime;
  double seq_time;
  FILE* log_file;

  double sum_parallel = 0;

  if (argc == 2) {
    // pass
  }
  else {
    printf("Usage: %s n x\n  where n is problem size\n", argv[0]);
    return 0;
  }

  THREADS = 2*2*2*2*2*2; // 2^3n
  BLOCKS = 2*2*2*2*2*2*2*2*2*2; // 2^5n
  NUM_VALS = THREADS*BLOCKS;
 
  // Initialize arr
  N = atoi(argv[1]);
  fakeN = nearestPowerOfTwo(N);
  arr = (int *) malloc(fakeN * sizeof(int));
  init(arr, fakeN);
  rng(arr, N);

  writeToFile("data/input.txt", arr, N);
  log_file = fopen("output/log.txt", "a");
  if (log_file == NULL) {
    printf("Error: can't open/create file");
    exit(1);
  }

  time_t rawtime;
  struct tm* timeinfo;
  time(&rawtime);
  timeinfo = localtime(&rawtime);
  printf("-----------------------------------------\n");
  printf("%s", asctime(timeinfo));
  printf("Problem Size: %d\n", N);
  printf("Process: %d\n", num_thread);
  fprintf(log_file, "-----------------------------------------\n");
  fprintf(log_file, "%s", asctime(timeinfo));
  fprintf(log_file, "Problem Size: %d\n", N);
  fprintf(log_file, "THREADS x BLOCKS: %d x %d\n", THREAD, BLOCKS);

  int* newArr;
  newArr = (int *) malloc(fakeN * sizeof(int));
  memcpy(newArr, arr, fakeN);

  // [Start Time]
  gettimeofday (&startwtime, NULL);

  parBitonicSort(newArr, fakeN);

  gettimeofday (&endwtime, NULL);
  // [End Time]


  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
          + endwtime.tv_sec - startwtime.tv_sec);
  seq_time *= 1000000;

  printf("Parallel wall clock time (microseconds) = %f\n", seq_time);
  fprintf(log_file, "Parallel wall clock time (microseconds) = %f\n", seq_time);

  writeToFile("data/output.txt", newArr, N);
      
  test(newArr, fakeN);
  sum_parallel += seq_time;
  free(newArr);
  

  printf("-----------------------------------------\n");
  fprintf(log_file, "-----------------------------------------\n\n\n");
  fclose(log_file);
  

  return 0;
}

/** -------------- SUB-PROCEDURES  ----------------- **/ 
void init(int* arr, int n) {
  int i = 0;
  for (i = 0; i < n; i++) {
    arr[i] = MAX_INT;
  }
}

void writeToFile(char* filename, int* arr, int n) {
  FILE* write = fopen(filename, "w");
  if (write != NULL) {
    int j;
    for (j = 0; j < n; j++)
      fprintf(write, "%d\n", arr[j]);
    fclose(write);
  }
}

void test(int* arr, int n) {
  int pass = 1;
  int i;
  for (i = 1; i < n; i++) {
    pass &= (arr[i-1] <= arr[i]);
  }

  printf("TEST %s\n",(pass) ? "PASSED" : "FAILED");
}

void rng(int* arr, int n) {
  int seed = 13515057;
  srand(seed);
  int i;
  for (i = 0; i < n; i++) {
    arr[i] = (int)rand();
  }
}

void swap(int* a, int* b) {
  int t;
  t = *a;
  *a = *b;
  *b = t;
}

void compare(int* arr, int i, int j, int dir) {
  if (dir == (arr[i] > arr[j])) 
    swap(&(arr[i]), &(arr[j]));
}

void parBitonicSort(int* arr) {
  float *arrValues;
  size_t size = NUM_VALS * sizeof(int);

  hipMalloc((void**) &arrValues, size);
  hipMemcpy(arrValues, values, size, hipMemcpyHostToDevice);

  dim3 blocks(BLOCKS, 1);    /* Number of blocks   */
  dim3 threads(THREADS, 1);  /* Number of threads  */

  int j, k;
  /* Major step */
  for (k = 2; k <= NUM_VALS; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      bitonicSortStep<<<blocks, threads>>>(arrValues, j, k);
    }
  }
  hipMemcpy(values, arrValues, size, hipMemcpyDeviceToHost);
  hipFree(arrValues);
}

int nearestPowerOfTwo(int x) {
  int i = 2;
  while (i < x) {
    i *= 2;
  }

  return i;
}
